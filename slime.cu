#include "hip/hip_runtime.h"
#include "slime.cuh"

__device__
float2 operator+(float2 a, float2 b)
{
  float2 out;
  out.x = a.x + b.x;
  out.y = a.y + b.y;
  return out;
}

__device__
float2 operator*(float2 a, float b)
{
  float2 out;
  out.x = a.x*b;
  out.y = a.y*b;
  return out;
}

__device__
uint hash(uint state)
{
  state ^= 2747636419u;
  state *= 2654435769u;
  state ^= state >> 16;
  state *= 2654435769u;
  state ^= state >> 16;
  state *= 2654435769u;
  return state;
}

__global__
void senseMap(uint n, struct TrailMap *trailMap)
{
  static int sensorSize = 2;

  uint index = blockIdx.x * blockDim.x + threadIdx.x;
  uint stride = blockDim.x * gridDim.x;
  for (uint i = index; i < n; i += stride)
  {
    int2 sensorCentre;
    sensorCentre.x = trailMap[i].x;
    sensorCentre.y = trailMap[i].y;

    uint sum = 0;

    for (int offsetX = -sensorSize; offsetX <= sensorSize; offsetX++)
    {
      for (int offsetY = -sensorSize; offsetY <= sensorSize; offsetY++)
      {
        int2 pos;
        pos.x = sensorCentre.x + offsetX;
        pos.y = sensorCentre.y + offsetY;

        if(pos.x >= 0 && pos.x < WINDOW_WIDTH && pos.y >= 0 && pos.y < WINDOW_HEIGHT)
        {
          sum += trailMap[pos.y * WINDOW_WIDTH + pos.x].val;
        }
      }
    }

    trailMap[i].sense = sum;
  }
}

__device__
uint sense(struct Agent *agent, float sensorAngleOffset, struct TrailMap *trailMap)
{
  static int sensorOffsetDst = 20;

  float sensorAngle = agent->angle + sensorAngleOffset;
  float2 sensorDir;
  sensorDir.x = cosf(sensorAngle);
  sensorDir.y = sinf(sensorAngle);
  int2 sensorCentre;
  sensorCentre.x = agent->position.x + sensorDir.x * sensorOffsetDst;
  sensorCentre.y = agent->position.y + sensorDir.y * sensorOffsetDst;
  
  uint sum;

  if(sensorCentre.x >= 0 && sensorCentre.x < WINDOW_WIDTH && sensorCentre.y >= 0 && sensorCentre.y < WINDOW_HEIGHT)
  {
    sum = trailMap[sensorCentre.y * WINDOW_WIDTH + sensorCentre.x].sense;
  }

  return sum;
}

__global__
void update(uint n, struct Agent *agents, struct TrailMap *trailMap)
{
  static float sensorAngleSpacing = (float) (30 * M_PI/180);

  uint index = blockIdx.x * blockDim.x + threadIdx.x;
  uint stride = blockDim.x * gridDim.x;
  for (uint i = index; i < n; i += stride)
  {
    Agent agent = agents[i];

    uint random = hash(agent.position.y * WINDOW_WIDTH + agent.position.x + hash(i % WINDOW_WIDTH));
  
    uint weightForward = sense(&agent, 0, trailMap);
    uint weightLeft = sense(&agent, sensorAngleSpacing, trailMap);
    uint weightRight = sense(&agent, -sensorAngleSpacing, trailMap);

    float randomSteerStrength = random/4294967295.0;

    if (weightForward > weightLeft && weightForward > weightRight)
    {
      agents[i].angle += 0;
    }
    else if (weightForward < weightLeft && weightForward < weightRight)
    {
      agents[i].angle += (randomSteerStrength - 0.5) * 2 * TURN_SPEED * DELTA_TIME;
    }
    else if (weightRight > weightLeft)
    {
      agents[i].angle -= randomSteerStrength * TURN_SPEED * DELTA_TIME;
    }
    else if (weightLeft > weightRight)
    {
      agents[i].angle += randomSteerStrength * TURN_SPEED * DELTA_TIME;
    }

    float2 direction, newPos;
    direction.x = cosf(agent.angle);
    direction.y = sinf(agent.angle);

    newPos = agent.position + direction * MOVE_SPEED * DELTA_TIME;

    if(newPos.x < 0 || newPos.x >= WINDOW_WIDTH || newPos.y < 0 || newPos.y >= WINDOW_HEIGHT)
    {
      newPos.x = min(WINDOW_WIDTH-0.01f, max(0.f, newPos.x));
      newPos.y = min(WINDOW_HEIGHT-0.01f, max(0.f, newPos.y));
      agents[i].angle = (random/4294967295.0) * 2 * M_PI;
    }

    agents[i].position = newPos;
    trailMap[(uint) newPos.y * WINDOW_WIDTH + (uint) newPos.x].val = 255;
  }
}

__global__
void processTrailMap(uint n, struct TrailMap *trailMap)
{
  uint index = blockIdx.x * blockDim.x + threadIdx.x;
  uint stride = blockDim.x * gridDim.x;
  for (uint i = index; i < n; i += stride)
  {
    float originalValue = trailMap[i].val;

    float sum = 0;
    for (int offsetX = -1; offsetX <= 1; offsetX++)
    {
      for (int offsetY = -1; offsetY <= 1; offsetY++)
      {
        int sampleX = trailMap[i].x + offsetX;
        int sampleY = trailMap[i].y + offsetY;

        if(sampleX >= 0 && sampleX < WINDOW_WIDTH && sampleY >= 0 && sampleY < WINDOW_HEIGHT)
        {
          sum += trailMap[i].val;
        }

        float blurResult = sum / 9;

        float alpha = min(1.0f, DIFFUSE_SPEED * DELTA_TIME);
        float diffusedValue = originalValue*(1-alpha) + blurResult*alpha;
        float diffusedAndEvaporatedValue = max(0.f, diffusedValue - EVAPORATE_SPEED * DELTA_TIME);

        trailMap[i].val = diffusedAndEvaporatedValue;
      }
    }
  }
}

__global__
void setPixels(uint n, struct TrailMap *trailMap, sf::Uint8 *pixels)
{
  uint index = blockIdx.x * blockDim.x + threadIdx.x;
  uint stride = blockDim.x * gridDim.x;
  for (uint i = index; i < n; i += stride)
  {
    pixels[4*i] = pow(trailMap[i].val,3)/(255*255)*0.8;
    pixels[4*i+1] = pow(trailMap[i].val,2)/255*0.8;
    pixels[4*i+2] = trailMap[i].val*0.6;
    pixels[4*i+3] = 255;
  }
}

void CUDA::wrapper(uint n, struct Agent *agents, struct TrailMap *trailMap, sf::Uint8 *pixels)
{
  // Run kernel on 1M elements on the GPU
  int blockSize = 256;
  int numBlocks = (n + blockSize - 1) / blockSize;
  senseMap<<<numBlocks, blockSize>>>(WINDOW_WIDTH*WINDOW_HEIGHT, trailMap);

  update<<<numBlocks, blockSize>>>(n, agents, trailMap);

  processTrailMap<<<numBlocks, blockSize>>>(WINDOW_WIDTH*WINDOW_HEIGHT, trailMap);

  setPixels<<<numBlocks, blockSize>>>(WINDOW_WIDTH*WINDOW_HEIGHT, trailMap, pixels);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
}